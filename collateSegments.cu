/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void collateSegments_gpu(int * src, int * scanResult, int * output, int numEdges) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int total_thread = blockDim.x * gridDim.x;
	int i ;
	for(i = tid; i < numEdges; i += total_thread)
		if(i == numEdges - 1 || src[i] != src[i + 1])
		output[src[i]] = scanResult[i];
}
