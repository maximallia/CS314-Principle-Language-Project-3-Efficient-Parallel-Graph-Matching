/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void strongestNeighborScan_gpu(int * src, int * oldDst, int * newDst, int * oldWeight, int * newWeight, int * madeChanges, int distance, int numEdges){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int total_thread = blockDim.x * gridDim.x;

    int i;
    for(i = tid; i < numEdges; i += total_thread){
        if( i < distance){
            newDst[i] = oldDst[i];
            newWeight[i] = oldWeight[i];
        }
        else{
            if(src[i] != src[i - distance]){
                newDst[i] = oldDst[i];
                newWeight[i] = oldWeight[i];
            }
            else{
                if(oldWeight[i] < oldWeight[i - distance]){
                    newDst[i] = oldDst[i- distance];
                    newWeight[i] = oldWeight[i - distance];
                }
                else if (oldWeight[i] == oldWeight[i - distance]){
                    if(oldDst[i] > oldDst[i - distance]){
                        newDst[i] = oldDst[i-distance];
                        newWeight[i] = oldWeight[i-distance];
                    }
                    else{
                        newDst[i] = oldDst[i];
                        newWeight[i] = oldWeight[i];
                    }
                }
                else{
                    newDst[i] = oldDst[i];
                    newWeight[i] = oldWeight[i];
                }
            if(oldDst[i] != newDst[i] || oldWeight[i] != newWeight[i])
             *madeChanges = 1;
	   }
        }
    }
}
